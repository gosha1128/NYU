#include "hip/hip_runtime.h"
/*

  wrapper to Alex's NVMatrix::max

  nvMax(in,axis,out)

  in is a mxn matrix
  out is a mx1 or 1xn vector
  axes is 1,2 depending on which dimension the max is taken over

  Note that I am not compiling through "make"
  I have to change the name of all C code with CUDA to ".cu" (instead of cpp)
  I am manually compiling using nvmex which is a wrapper for mex that can handle .cu


MATLAB=/opt/pkg/matlab/current ./nvmex -f nvopts.sh  -DUNIX -outdir . nvMax.cu nvmatrix.cu nvmatrix_kernel.cu ../../common/GPUmat.cpp  -I"/usr/local/pkg/cuda/3.1/cuda/include" -I"../../include"  -L"/usr/local/pkg/cuda/3.1/cuda/lib64" -lcuda -lcudart -lcufft -lcublas

Note that I am passing in both this file and the .cu code for the kernel (nvmatrix_kernel.cu) -- otherwise I will get undefined symbols

This document was useful:
http://faculty.washington.edu/dushaw/epubs/Matlab_CUDA_Tutorial_2_10.pdf

nvmex, nvopts.sh came from here: 
http://developer.nvidia.com/object/matlab_cuda.html
 */
#include <stdio.h>
#include <string.h>
#include <stdarg.h>

#ifdef UNIX
#include <stdint.h>
#endif

#include "mex.h"

// CUDA
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "GPUmat.hh"
#include "GPUkernel.hh"

//#include "misc.cuh"
#include "nvmatrix.cuh"

//#include "nvmatrix_kernel.cuh"

// static paramaters
//static hipFunction_t drvfunf; // float
//static hipFunction_t drvfunf2; // float

//static hipFunction_t drvfunc; // complex
//static hipFunction_t drvfund; // double
//static hipFunction_t drvfuncd;//double complex


static int init = 0;

static GPUmat *gm;

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {

  //hipError_t cudastatus = hipSuccess;

  if (nrhs != 3) 
    mexErrMsgTxt("Wrong number of arguments");

  if (init == 0) {
    // Initialize function
    //mexLock();

    // load GPUmat
    gm = gmGetGPUmat();

    init = 1;
  }

  // mex parameters are:

  // 1. IN1 (in)
  // 2. IN2 (axis) - {1,2}
  // 2. OUT (out)

  //IN1 is an input GPU array
  GPUtype IN1 = gm->gputype.getGPUtype(prhs[0]);
  gpuTYPE_t tIn1 = gm->gputype.getType(IN1);

  //IN2 is an axis {1,2}
  //Note it is converted to C-style indexing {0,1}
  int axis = (int) mxGetScalar(prhs[1]) - 1;

  //OUT is the output GPU array
  GPUtype OUT = gm->gputype.getGPUtype(prhs[2]);
  gpuTYPE_t tOut = gm->gputype.getType(OUT);

  //dimensions
  const int * sIn1 = gm->gputype.getSize(IN1);
  const int * sOut = gm->gputype.getSize(OUT);


  // if (&prhs[1] == &prhs[2])
  //   mexErrMsgTxt("vec and target cannot be the same");

  // if (sIn2[0] !=1 && sIn2[1] !=1)
  //   mexErrMsgTxt("second argument is not a vector");

  // if (sIn2[0] != sIn1[0] && sIn2[1] != sIn1[1])
  //   mexErrMsgTxt("vec must match input in EITHER rows OR cols");

  // if (sOut[0] != sIn1[0] || sOut[1] != sIn1[1])
  //   mexErrMsgTxt("target dims must match input dims");


  // /* 
  //    Modified Alex's kernels to support col-major data
  //    So these are now defined properly
  // */
  // const unsigned int height = sIn1[0];
  // const unsigned int width = sIn1[1];

  // // Output for debugging
  // //mexPrintf("input rows: %d cols: %d\n",sIn1[0],sIn1[1]);
  // //mexPrintf("vec rows: %d cols: %d\n",sIn2[0],sIn2[1]);
  // //mexPrintf("output: %d x %d\n",sOut[0],sOut[1]);
  // //mexPrintf("wxh: %d x %d\n",width,height);

  // hipFunction_t drvfun;
  // hipFunction_t drvfun2;
  // if ((tIn1 == gpuFLOAT) && (tIn2 == gpuFLOAT)) {
  //   drvfun = drvfunf;
  //   drvfun2 = drvfunf2;
  // }
  // else {
  //   mexErrMsgTxt("Only singles are supported at present.");
  // }

  // I need the pointers to GPU memory
  hipDeviceptr_t d_IN1  = (hipDeviceptr_t) (UINTPTR gm->gputype.getGPUptr(IN1));
  // hipDeviceptr_t d_IN2  = (hipDeviceptr_t) (UINTPTR gm->gputype.getGPUptr(IN2));
  hipDeviceptr_t d_OUT = (hipDeviceptr_t) (UINTPTR gm->gputype.getGPUptr(OUT));

  /* Create NVMatrix by initializing to device data in (in)
     I think the right thing to do is pass in true for the isTrans parameter
     Since GPUsingle data is in col-major rather than row-major order */
  NVMatrix nvIn((float*) d_IN1,sIn1[0],sIn1[1],true);
  NVMatrix nvOut((float*) d_OUT,sOut[0],sOut[1],true);

  nvIn.max(axis,nvOut);
  
}
