#include "hip/hip_runtime.h"
#include "GPUkernel.hh"

typedef float2  Complex;
typedef double2 DoubleComplex;

__device__ inline float times(float data1, float data2) {
  return data1*data2;
}
__device__ inline Complex times(Complex data1, Complex data2) {
  return make_float2(data1.x * data2.x - data1.y * data2.y, data1.x * data2.y + data1.y * data2.x);
}
__device__ inline double times(double data1, double data2) {
  return data1*data2;
}
__device__ inline DoubleComplex times(DoubleComplex data1, DoubleComplex data2) {
  return make_double2(data1.x * data2.x - data1.y * data2.y, data1.x * data2.y + data1.y * data2.x);
}

extern "C" {

/* PLUS FLOAT */
__global__ void PLUSF(int n, 
                      int offset, 
                      float * idata1, 
                      float * idata2, 
                      float * odata)
{  
    unsigned int xIndex = blockIdx.x * BLOCK_DIM1D + threadIdx.x + offset;
    if ((xIndex - offset) < n) 
        odata[xIndex] = idata1[xIndex] + idata2[xIndex];
}

/* PLUS COMPLEX FLOAT */
__global__ void PLUSC(int n, 
                      int offset, 
                      Complex * idata1, 
                      Complex * idata2, 
                      Complex * odata)
{  
    unsigned int xIndex = blockIdx.x * BLOCK_DIM1D + threadIdx.x + offset;
    if ((xIndex - offset) < n) { 
        Complex tmp = make_float2(0.0,0.0);
        Complex in1 = idata1[xIndex];
        Complex in2 = idata2[xIndex];
        tmp.x = in1.x + in2.x;
        tmp.y = in1.y + in2.y;
        odata[xIndex] = tmp;
    }
}

/* PLUS DOUBLE */
__global__ void PLUSD(int n, 
                      int offset, 
                      double * idata1, 
                      double * idata2, 
                      double * odata)
{  
    unsigned int xIndex = blockIdx.x * BLOCK_DIM1D + threadIdx.x + offset;
    if ((xIndex - offset) < n) 
        odata[xIndex] = idata1[xIndex] + idata2[xIndex];
}

/* PLUS COMPLEX DOUBLE */
__global__ void PLUSCD(int n, 
                      int offset, 
                      DoubleComplex * idata1, 
                      DoubleComplex * idata2, 
                      DoubleComplex * odata)
{  
    unsigned int xIndex = blockIdx.x * BLOCK_DIM1D + threadIdx.x + offset;
    if ((xIndex - offset) < n) { 
        DoubleComplex tmp = make_double2(0.0,0.0);
        DoubleComplex in1 = idata1[xIndex];
        DoubleComplex in2 = idata2[xIndex];
        tmp.x = in1.x + in2.x;
        tmp.y = in1.y + in2.y;
        odata[xIndex] = tmp;
    }
}

/******************************************************************************/

/* TIMES FLOAT */
__global__ void TIMESF(int n, 
                      int offset, 
                      float * idata1, 
                      float * idata2, 
                      float * odata)
{  
    unsigned int xIndex = blockIdx.x * BLOCK_DIM1D + threadIdx.x + offset;
    if ((xIndex - offset) < n) 
        odata[xIndex] = times(idata1[xIndex] , idata2[xIndex]);
}

/* TIMES COMPLEX FLOAT */
__global__ void TIMESC(int n, 
                      int offset, 
                      Complex * idata1, 
                      Complex * idata2, 
                      Complex * odata)
{  
    unsigned int xIndex = blockIdx.x * BLOCK_DIM1D + threadIdx.x + offset;
    if ((xIndex - offset) < n) { 
        odata[xIndex] = times(idata1[xIndex] , idata2[xIndex]);
    }
}

/* TIMES DOUBLE */


__global__ void TIMESD(int n, 
                      int offset, 
                      double * idata1, 
                      double * idata2, 
                      double * odata)
{  
    unsigned int xIndex = blockIdx.x * BLOCK_DIM1D + threadIdx.x + offset;
    if ((xIndex - offset) < n) 
        odata[xIndex] = times(idata1[xIndex], idata2[xIndex]);
}

/* TIMES COMPLEX DOUBLE */
__global__ void TIMESCD(int n, 
                      int offset, 
                      DoubleComplex * idata1, 
                      DoubleComplex * idata2, 
                      DoubleComplex * odata)
{  
    unsigned int xIndex = blockIdx.x * BLOCK_DIM1D + threadIdx.x + offset;
    if ((xIndex - offset) < n) { 
        odata[xIndex] = times(idata1[xIndex] , idata2[xIndex]);
    }
}


}
